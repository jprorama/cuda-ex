
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

// Device code
__global__ void VecAdd(float* A, float* B, float* C, int N)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < N)
    C[i] = A[i] + B[i];
}
  
float *initarray(float *a, int N, float value) {
  int i;

  for (i=0; i<N; i++)
      a[i] = drand48()*value;

  return a;
}

void printarray(float *a, int N) {
  int i;

  for (i=0; i<N; i++) {
      printf("%f ", a[i]);
    printf("\n");
  }
}
          
// Host code
int main(int argc, char **argv)
{
  int N = atoi(argv[1]);
  size_t size = N * sizeof(float);

  // Allocate input vectors h_A and h_B in host memory
  float* h_A = (float*)malloc(size);
  float* h_B = (float*)malloc(size);
  float* h_C = (float*)malloc(size);
  
  // Initialize input vectors
  initarray(h_A, N, 10);
  initarray(h_B, N, 10);

  printarray(h_A, N);
  printarray(h_B, N);

  // Allocate vectors in device memory
  float* d_A;
  hipMalloc(&d_A, size);
  float* d_B;
  hipMalloc(&d_B, size);
  float* d_C;
  hipMalloc(&d_C, size);

  // Copy vectors from host memory to device memory
  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

  // Invoke kernel
  int threadsPerBlock = 256;
  int blocksPerGrid =
    (N + threadsPerBlock - 1) / threadsPerBlock;
  VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

  // Copy result from device memory to host memory
  // h_C contains the result in host memory
  hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

  // print result
  printarray(h_C, N);

  // Free device memory
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
            
  // Free host memory
  free(h_A);
  free(h_B);
  free(h_C);
}
