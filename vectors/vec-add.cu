
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

// Device code
__global__ void VecAdd(float* A, float* B, float* C, int N)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < N)
    C[i] = A[i] + B[i];
}
  
float *initarray(float *a, int N, float value) {
  int i;

  for (i=0; i<N; i++)
      a[i] = drand48()*value;

  return a;
}

void printarray(float *a, int N) {
  int i;

  for (i=0; i<N; i++) {
      printf("%f ", a[i]);
    printf("\n");
  }
}
          
// Host code
int main(int argc, char **argv)
{
  int N = atoi(argv[1]);
  size_t size = N * sizeof(float);

  // Allocate input vectors h_A and h_B in host memory
  float* h_A = (float*)malloc(size);
  float* h_B = (float*)malloc(size);
  float* h_C = (float*)malloc(size);
  
  // Initialize input vectors
  initarray(h_A, N, 10);
  initarray(h_B, N, 10);

#ifdef DEBUG
  printf("h_a:\n");
  printarray(h_A, N);
  printf("h_b:\n");
  printarray(h_B, N);
#endif

  // Allocate vectors in device memory
  float* d_A;
  hipMalloc(&d_A, size);
  float* d_B;
  hipMalloc(&d_B, size);
  float* d_C;
  hipMalloc(&d_C, size);

  // Copy vectors from host memory to device memory
  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

  // Invoke kernel
  int threadsPerBlock = 256;
  int blocksPerGrid =
    (N + threadsPerBlock - 1) / threadsPerBlock;
  VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

  // Copy result from device memory to host memory
  // h_C contains the result in host memory
  hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

#ifdef DEBUG
  // print result
  printf("h_c:\n");
  printarray(h_C, N);
#endif

  // Free device memory
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
            
  // Free host memory
  free(h_A);
  free(h_B);
  free(h_C);
}
